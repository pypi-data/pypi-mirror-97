#include "hip/hip_runtime.h"
/*
   Implements the Landau kernel
*/
#include <petscconf.h>
#include <petsc/private/dmpleximpl.h>   /*I  "dmpleximpl.h"   I*/
#include <petsclandau.h>
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1
#include <../src/mat/impls/aij/seq/aij.h>
#include <petsc/private/kernels/petscaxpy.h>

#define PETSC_THREAD_SYNC __syncthreads()
#define PETSC_DEVICE_FUNC_DECL __device__
#include "../land_kernel.h"

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err));        \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)
// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err));        \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err));       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

#define LANDAU_USE_SHARED_GPU_MEM
//j
// The GPU Landau kernel
//
__global__
void landau_kernel(const PetscInt nip, const PetscInt dim, const PetscInt totDim, const PetscInt Nf, const PetscInt Nb, const PetscReal invJj[],
                   const PetscReal nu_alpha[], const PetscReal nu_beta[], const PetscReal invMass[], const PetscReal Eq_m[],
                   const PetscReal * const BB, const PetscReal * const DD, const PetscReal * const IPDataGlobal, const PetscReal wiGlobal[],
#if !defined(LANDAU_USE_SHARED_GPU_MEM)
                   PetscReal *g2arr, PetscReal *g3arr,
#endif
                   PetscBool quarter3DDomain, PetscScalar elemMats_out[])
{
  const PetscInt  Nq = blockDim.x, myelem = blockIdx.x;
#if defined(LANDAU_USE_SHARED_GPU_MEM)
  extern __shared__ PetscReal g2_g3_qi[]; // Nq * { [NSubBlocks][Nf][dim] ; [NSubBlocks][Nf][dim][dim] }
  PetscReal       (*g2)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM]         = (PetscReal (*)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM])         &g2_g3_qi[0];
  PetscReal       (*g3)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM][LANDAU_DIM] = (PetscReal (*)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM][LANDAU_DIM]) &g2_g3_qi[LANDAU_MAX_SUB_THREAD_BLOCKS*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM];
#else
  PetscReal       (*g2)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM]         = (PetscReal (*)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM])         &g2arr[myelem*LANDAU_MAX_SUB_THREAD_BLOCKS*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM       ];
  PetscReal       (*g3)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM][LANDAU_DIM] = (PetscReal (*)[LANDAU_MAX_NQ][LANDAU_MAX_SUB_THREAD_BLOCKS][LANDAU_MAX_SPECIES][LANDAU_DIM][LANDAU_DIM]) &g3arr[myelem*LANDAU_MAX_SUB_THREAD_BLOCKS*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM*LANDAU_DIM];
#endif
  const PetscInt  myQi = threadIdx.x, mySubBlk = threadIdx.y, nSubBlks = blockDim.y;
  const PetscInt  jpidx = myQi + myelem * Nq;
  const PetscInt  subblocksz = nip/nSubBlks + !!(nip%nSubBlks), ip_start = mySubBlk*subblocksz, ip_end = (mySubBlk+1)*subblocksz > nip ? nip : (mySubBlk+1)*subblocksz; /* this could be wrong with very few global IPs */
  PetscScalar     *elemMat  = &elemMats_out[myelem*totDim*totDim]; /* my output */

  if (threadIdx.x==0 && threadIdx.y==0) {
    memset(elemMat, 0, totDim*totDim*sizeof(PetscScalar));
  }
  __syncthreads();
  landau_inner_integral(myQi, Nq, mySubBlk, nSubBlks, ip_start, ip_end, 1,        jpidx, Nf, dim, IPDataGlobal, wiGlobal, &invJj[jpidx*dim*dim], nu_alpha, nu_beta, invMass, Eq_m, quarter3DDomain, Nq, Nb, 0, Nq, BB, DD, elemMat, *g2, *g3, myelem); /* compact */
  // landau_inner_integral(myQi, Nq, mySubBlk, nSubBlks, mySubBlk,    nip, nSubBlks, jpidx, Nf, dim, IPDataGlobal, wiGlobal, &invJj[jpidx*dim*dim], nu_alpha, nu_beta, invMass, Eq_m, quarter3DDomain, Nq, Nb, 0, Nq, BB, DD, elemMat, *g2, *g3, myelem); /* spread */
}
static PetscErrorCode LandauAssembleCuda(PetscInt cStart, PetscInt cEnd, PetscInt totDim, DM plex, PetscSection section, PetscSection globalSection, Mat JacP, PetscScalar elemMats[], PetscContainer container, const PetscLogEvent events[]);
__global__ void assemble_kernel(const PetscInt nidx_arr[], PetscInt *idx_arr[], PetscScalar *el_mats[], const ISColoringValue colors[], Mat_SeqAIJ mats[]);
PetscErrorCode LandauCUDAJacobian(DM plex, const PetscInt Nq, const PetscReal nu_alpha[],const PetscReal nu_beta[],
                                  const PetscReal invMass[], const PetscReal Eq_m[], const PetscReal * const IPDataGlobal,
                                  const PetscReal wiGlobal[], const PetscReal invJj[], const PetscInt num_sub_blocks, const PetscLogEvent events[], PetscBool quarter3DDomain,
                                  Mat JacP)
{
  PetscErrorCode    ierr;
  PetscInt          ii,ej,*Nbf,Nb,nip_dim2,cStart,cEnd,Nf,dim,numGCells,totDim,nip,szf=sizeof(PetscReal);
  PetscReal         *d_BB,*d_DD,*d_invJj,*d_wiGlobal,*d_nu_alpha,*d_nu_beta,*d_invMass,*d_Eq_m;
  PetscScalar       *elemMats,*d_elemMats;
  PetscLogDouble    flops;
  PetscTabulation   *Tf;
  PetscDS           prob;
  PetscSection      section, globalSection;
  PetscReal        *d_IPDataGlobal;
  PetscBool         cuda_assemble = PETSC_FALSE;
  PetscFunctionBegin;

  ierr = PetscLogEventBegin(events[3],0,0,0,0);CHKERRQ(ierr);
  ierr = DMGetDimension(plex, &dim);CHKERRQ(ierr);
  if (dim!=LANDAU_DIM) SETERRQ(PETSC_COMM_SELF, PETSC_ERR_PLIB, "LANDAU_DIM != dim");
  ierr = DMPlexGetHeightStratum(plex,0,&cStart,&cEnd);CHKERRQ(ierr);
  numGCells = cEnd - cStart;
  nip  = numGCells*Nq; /* length of inner global iteration */
  ierr = DMGetDS(plex, &prob);CHKERRQ(ierr);
  ierr = PetscDSGetNumFields(prob, &Nf);CHKERRQ(ierr);
  ierr = PetscDSGetDimensions(prob, &Nbf);CHKERRQ(ierr); Nb = Nbf[0];
  if (Nq != Nb) SETERRQ2(PETSC_COMM_SELF, PETSC_ERR_PLIB, "Nq != Nb. %D  %D",Nq,Nb);
  ierr = PetscDSGetTotalDimension(prob, &totDim);CHKERRQ(ierr);
  ierr = PetscDSGetTabulation(prob, &Tf);CHKERRQ(ierr);
  ierr = DMGetLocalSection(plex, &section);CHKERRQ(ierr);
  ierr = DMGetGlobalSection(plex, &globalSection);CHKERRQ(ierr);
  // create data
  CUDA_SAFE_CALL(hipMalloc((void **)&d_IPDataGlobal, nip*(dim + Nf*(dim+1))*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_nu_alpha, Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_nu_beta,  Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_invMass,  Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMalloc((void **)&d_Eq_m,     Nf*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(d_IPDataGlobal, IPDataGlobal, nip*(dim + Nf*(dim+1))*szf, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_nu_alpha, nu_alpha, Nf*szf,                             hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_nu_beta,  nu_beta,  Nf*szf,                             hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_invMass,  invMass,  Nf*szf,                             hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_Eq_m,     Eq_m,     Nf*szf,                             hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_BB,              Nq*Nb*szf));     // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_BB, Tf[0]->T[0], Nq*Nb*szf,   hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_DD,              Nq*Nb*dim*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_DD, Tf[0]->T[1], Nq*Nb*dim*szf,   hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_wiGlobal,           Nq*numGCells*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_wiGlobal, wiGlobal, Nq*numGCells*szf,   hipMemcpyHostToDevice));
  // collect geometry
  flops = (PetscLogDouble)numGCells*(PetscLogDouble)Nq*(PetscLogDouble)(5.*dim*dim*Nf*Nf + 165.);
  nip_dim2 = Nq*numGCells*dim*dim;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_invJj, nip_dim2*szf)); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(d_invJj, invJj, nip_dim2*szf,       hipMemcpyHostToDevice));
  ierr = PetscLogEventEnd(events[3],0,0,0,0);CHKERRQ(ierr);

  ierr = PetscLogEventBegin(events[4],0,0,0,0);CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(flops*nip);CHKERRQ(ierr);
  {
    dim3 dimBlock(Nq,num_sub_blocks);
    CUDA_SAFE_CALL(hipMalloc((void **)&d_elemMats, totDim*totDim*numGCells*sizeof(PetscScalar))); // kernel output
    ii = LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM*(1+LANDAU_DIM)*LANDAU_MAX_SUB_THREAD_BLOCKS;
#if defined(LANDAU_USE_SHARED_GPU_MEM)
    // PetscPrintf(PETSC_COMM_SELF,"Call land_kernel with %D kB shared memory\n",ii*8/1024);
    landau_kernel<<<numGCells,dimBlock,ii*szf>>>(nip,dim,totDim,Nf,Nb,d_invJj,d_nu_alpha,d_nu_beta,d_invMass,d_Eq_m,
                                                 d_BB, d_DD, d_IPDataGlobal, d_wiGlobal, quarter3DDomain, d_elemMats);
    CHECK_LAUNCH_ERROR();
#else
    PetscReal  *d_g2g3;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_g2g3, ii*szf*numGCells)); // kernel input
    PetscReal  *g2 = &d_g2g3[0];
    PetscReal  *g3 = &d_g2g3[LANDAU_MAX_SUB_THREAD_BLOCKS*LANDAU_MAX_NQ*LANDAU_MAX_SPECIES*LANDAU_DIM*numGCells];
    landau_kernel<<<numGCells,dimBlock>>>(nip,dim,totDim,Nf,Nb,d_invJj,d_nu_alpha,d_nu_beta,d_invMass,d_Eq_m,
                                          d_BB, d_DD, d_IPDataGlobal, d_wiGlobal, g2, g3, quarter3DDomain, d_elemMats);
    CHECK_LAUNCH_ERROR();
    CUDA_SAFE_CALL (hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipFree(d_g2g3));
#endif
  }
  ierr = PetscLogEventEnd(events[4],0,0,0,0);CHKERRQ(ierr);
  // delete device data
  ierr = PetscLogEventBegin(events[5],0,0,0,0);CHKERRQ(ierr);
  CUDA_SAFE_CALL(hipFree(d_IPDataGlobal));
  CUDA_SAFE_CALL(hipFree(d_invJj));
  CUDA_SAFE_CALL(hipFree(d_wiGlobal));
  CUDA_SAFE_CALL(hipFree(d_nu_alpha));
  CUDA_SAFE_CALL(hipFree(d_nu_beta));
  CUDA_SAFE_CALL(hipFree(d_invMass));
  CUDA_SAFE_CALL(hipFree(d_Eq_m));
  CUDA_SAFE_CALL(hipFree(d_BB));
  CUDA_SAFE_CALL(hipFree(d_DD));
  ierr = PetscMalloc1(totDim*totDim*numGCells,&elemMats);CHKERRQ(ierr);
  CUDA_SAFE_CALL(hipMemcpy(elemMats, d_elemMats, totDim*totDim*numGCells*sizeof(PetscScalar), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(d_elemMats));
  ierr = PetscLogEventEnd(events[5],0,0,0,0);CHKERRQ(ierr);

  ierr = PetscLogEventBegin(events[6],0,0,0,0);CHKERRQ(ierr);
  if (!cuda_assemble) {
    PetscScalar *elMat;
    for (ej = cStart, elMat = elemMats ; ej < cEnd; ++ej, elMat += totDim*totDim) {
      ierr = DMPlexMatSetClosure(plex, section, globalSection, JacP, ej, elMat, ADD_VALUES);CHKERRQ(ierr);
      if (ej==-1) {
        int d,f;
        printf("GPU Element matrix\n");
        for (d = 0; d < totDim; ++d) {
          for (f = 0; f < totDim; ++f) printf(" %17.10e",  PetscRealPart(elMat[d*totDim + f]));
          printf("\n");
        }
        exit(12);
      }
    }
  } else {
    PetscContainer container = NULL;
    ierr = PetscObjectQuery((PetscObject)JacP,"coloring",(PetscObject*)&container);CHKERRQ(ierr);
    if (!container) {
      ierr = PetscLogEventBegin(events[8],0,0,0,0);CHKERRQ(ierr);
      ierr = LandauCreateColoring(JacP, plex, &container);CHKERRQ(ierr);
      ierr = PetscLogEventEnd(events[8],0,0,0,0);CHKERRQ(ierr);
    }
    ierr = LandauAssembleCuda(cStart, cEnd, totDim, plex, section, globalSection, JacP, elemMats, container, events);CHKERRQ(ierr);
  }
  ierr = PetscFree(elemMats);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(events[6],0,0,0,0);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}

__global__
void assemble_kernel(const PetscInt nidx_arr[], PetscInt *idx_arr[], PetscScalar *el_mats[], const ISColoringValue colors[], Mat_SeqAIJ mats[])
{
  const PetscInt     myelem = (gridDim.x==1) ? threadIdx.x : blockIdx.x;
  Mat_SeqAIJ         a = mats[colors[myelem]]; /* copy to GPU */
  const PetscScalar *v = el_mats[myelem];
  const PetscInt    *in = idx_arr[myelem], *im = idx_arr[myelem], n = nidx_arr[myelem], m = nidx_arr[myelem];
  /* mat set values */
  PetscInt          *rp,k,low,high,t,row,nrow,i,col,l;
  PetscInt          *ai = a.i,*ailen = a.ilen;
  PetscInt          *aj = a.j,lastcol = -1;
  MatScalar         *ap=NULL,value=0.0,*aa = a.a;
  for (k=0; k<m; k++) { /* loop over added rows */
    row = im[k];
    if (row < 0) continue;
    rp   = aj + ai[row];
    ap = aa + ai[row];
    nrow = ailen[row];
    low  = 0;
    high = nrow;
    for (l=0; l<n; l++) { /* loop over added columns */
      /* if (in[l] < 0) { */
      /*   printf("\t\tin[l] < 0 ?????\n"); */
      /*   continue; */
      /* } */
      while (l<n && (value = v[l + k*n], PetscAbsScalar(value)==0.0)) l++;
      if (l==n) break;
      col = in[l];
      if (col <= lastcol) low = 0;
      else high = nrow;
      lastcol = col;
      while (high-low > 5) {
        t = (low+high)/2;
        if (rp[t] > col) high = t;
        else low = t;
      }
      for (i=low; i<high; i++) {
        // if (rp[i] > col) break;
        if (rp[i] == col) {
          ap[i] += value;
          low = i + 1;
          goto noinsert;
        }
      }
      printf("\t\t\t ERROR in assemble_kernel\n");
    noinsert:;
    }
  }
}

static PetscErrorCode LandauAssembleCuda(PetscInt cStart, PetscInt cEnd, PetscInt totDim, DM plex, PetscSection section, PetscSection globalSection, Mat JacP, PetscScalar elemMats[], PetscContainer container, const PetscLogEvent events[])
{
  PetscErrorCode    ierr;
#define LANDAU_MAX_COLORS 16
#define LANDAU_MAX_ELEMS 512
  Mat_SeqAIJ             h_mats[LANDAU_MAX_COLORS], *jaca = (Mat_SeqAIJ *)JacP->data, *d_mats;
  const PetscInt         nelems = cEnd - cStart, nnz = jaca->i[JacP->rmap->n], N = JacP->rmap->n;  /* serial */
  const ISColoringValue *colors;
  ISColoringValue       *d_colors,colour;
  PetscInt              *h_idx_arr[LANDAU_MAX_ELEMS], h_nidx_arr[LANDAU_MAX_ELEMS], *d_nidx_arr, **d_idx_arr,nc,ej,j,cell;
  PetscScalar           *h_new_el_mats[LANDAU_MAX_ELEMS], *val_buf, **d_new_el_mats;
  ISColoring             iscoloring;
  ierr = PetscContainerGetPointer(container,(void**)&iscoloring);CHKERRQ(ierr);
  /* get colors */
  ierr = ISColoringGetColors(iscoloring, &j, &nc, &colors);CHKERRQ(ierr);
  if (nelems>LANDAU_MAX_ELEMS) SETERRQ2(PETSC_COMM_SELF, PETSC_ERR_PLIB, "too many elements. %D > %D",nelems,LANDAU_MAX_ELEMS);
  if (nc>LANDAU_MAX_COLORS) SETERRQ2(PETSC_COMM_SELF, PETSC_ERR_PLIB, "too many colors. %D > %D",nc,LANDAU_MAX_COLORS);
  /* colors for kernel */
  CUDA_SAFE_CALL(hipMalloc((void **)&d_colors,         nelems*sizeof(ISColoringValue))); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_colors, colors, nelems*sizeof(ISColoringValue), hipMemcpyHostToDevice));
  /* get indices and element matrices */
  for (cell = cStart, ej = 0 ; cell < cEnd; ++cell, ++ej) {
    PetscInt numindices,*indices;
    PetscScalar *elMat = &elemMats[ej*totDim*totDim];
    PetscScalar *valuesOrig = elMat;
    ierr = DMPlexGetClosureIndices(plex, section, globalSection, cell, PETSC_TRUE, &numindices, &indices, NULL, (PetscScalar **) &elMat);CHKERRQ(ierr);
    h_nidx_arr[ej] = numindices;
    CUDA_SAFE_CALL(hipMalloc((void **)&h_idx_arr[ej],            numindices*sizeof(PetscInt))); // kernel input
    CUDA_SAFE_CALL(hipMemcpy(          h_idx_arr[ej],   indices, numindices*sizeof(PetscInt), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void **)&h_new_el_mats[ej],        numindices*numindices*sizeof(PetscScalar))); // kernel input
    CUDA_SAFE_CALL(hipMemcpy(          h_new_el_mats[ej], elMat, numindices*numindices*sizeof(PetscScalar), hipMemcpyHostToDevice));
    ierr = DMPlexRestoreClosureIndices(plex, section, globalSection, cell, PETSC_TRUE, &numindices, &indices, NULL, (PetscScalar **) &elMat);CHKERRQ(ierr);
    if (elMat != valuesOrig) {ierr = DMRestoreWorkArray(plex, numindices*numindices, MPIU_SCALAR, &elMat);CHKERRQ(ierr);}
  }
  CUDA_SAFE_CALL(hipMalloc((void **)&d_nidx_arr,                  nelems*sizeof(PetscInt))); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_nidx_arr,    h_nidx_arr,   nelems*sizeof(PetscInt), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_idx_arr,                   nelems*sizeof(PetscInt*))); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_idx_arr,     h_idx_arr,    nelems*sizeof(PetscInt*), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_new_el_mats,               nelems*sizeof(PetscScalar*))); // kernel input
  CUDA_SAFE_CALL(hipMemcpy(          d_new_el_mats, h_new_el_mats,nelems*sizeof(PetscScalar*), hipMemcpyHostToDevice));
  /* make matrix buffers */
  for (colour=0; colour<nc; colour++) {
    Mat_SeqAIJ *a = &h_mats[colour];
    /* create on GPU and copy to GPU */
    CUDA_SAFE_CALL(hipMalloc((void **)&a->i,               (N+1)*sizeof(PetscInt))); // kernel input
    CUDA_SAFE_CALL(hipMemcpy(          a->i,    jaca->i,   (N+1)*sizeof(PetscInt), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void **)&a->ilen,            (N)*sizeof(PetscInt))); // kernel input
    CUDA_SAFE_CALL(hipMemcpy(          a->ilen, jaca->ilen,(N)*sizeof(PetscInt), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void **)&a->j,               (nnz)*sizeof(PetscInt))); // kernel input
    CUDA_SAFE_CALL(hipMemcpy(          a->j,    jaca->j,   (nnz)*sizeof(PetscInt), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void **)&a->a,               (nnz)*sizeof(PetscScalar))); // kernel output
    CUDA_SAFE_CALL(hipMemset(          a->a, 0,            (nnz)*sizeof(PetscScalar)));
  }
  CUDA_SAFE_CALL(hipMalloc(&d_mats,         nc*sizeof(Mat_SeqAIJ))); // kernel input
  CUDA_SAFE_CALL(hipMemcpy( d_mats, h_mats, nc*sizeof(Mat_SeqAIJ), hipMemcpyHostToDevice));
  /* do it */
  assemble_kernel<<<nelems,1>>>(d_nidx_arr, d_idx_arr, d_new_el_mats, d_colors, d_mats);
  CHECK_LAUNCH_ERROR();
  /* cleanup */
  CUDA_SAFE_CALL(hipFree(d_colors));
  CUDA_SAFE_CALL(hipFree(d_nidx_arr));
  for (ej = cStart ; ej < nelems; ++ej) {
    CUDA_SAFE_CALL(hipFree(h_idx_arr[ej]));
    CUDA_SAFE_CALL(hipFree(h_new_el_mats[ej]));
  }
  CUDA_SAFE_CALL(hipFree(d_idx_arr));
  CUDA_SAFE_CALL(hipFree(d_new_el_mats));
  /* copy & add Mat data back to CPU to JacP */

  ierr = PetscLogEventBegin(events[2],0,0,0,0);CHKERRQ(ierr);
  ierr = PetscMalloc1(nnz,&val_buf);CHKERRQ(ierr);
  ierr = PetscMemzero(jaca->a,nnz*sizeof(PetscScalar));CHKERRQ(ierr);
  for (colour=0; colour<nc; colour++) {
    Mat_SeqAIJ *a = &h_mats[colour];
    CUDA_SAFE_CALL(hipMemcpy(val_buf, a->a, (nnz)*sizeof(PetscScalar), hipMemcpyDeviceToHost));
    PetscKernelAXPY(jaca->a,1.0,val_buf,nnz);
  }
  ierr = PetscFree(val_buf);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(events[2],0,0,0,0);CHKERRQ(ierr);

  for (colour=0; colour<nc; colour++) {
    Mat_SeqAIJ *a = &h_mats[colour];
    /* destroy mat */
    CUDA_SAFE_CALL(hipFree(a->i));
    CUDA_SAFE_CALL(hipFree(a->ilen));
    CUDA_SAFE_CALL(hipFree(a->j));
    CUDA_SAFE_CALL(hipFree(a->a));
  }
  CUDA_SAFE_CALL(hipFree(d_mats));
  PetscFunctionReturn(0);
}
