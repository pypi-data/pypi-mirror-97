#include "hip/hip_runtime.h"
/*
 * Copyright Alex Yu 2021
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdint>
#include "common.cuh"
#include "data_spec_packed.cuh"

#define CUDA_N_THREADS 1024

namespace {
void check_indices(torch::Tensor& indices) {
    CHECK_INPUT(indices);
    TORCH_CHECK(indices.dim() == 2);
    TORCH_CHECK(indices.is_floating_point());
}

namespace device {

template <typename scalar_t>
__device__ __inline__ scalar_t* get_tree_leaf_ptr(
       torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits>
        data,
       PackedTreeSpec<scalar_t>& __restrict__ tree,
       const scalar_t* __restrict__ xyz_ind,
       int32_t* node_id) {
    scalar_t xyz[3] = {xyz_ind[0], xyz_ind[1], xyz_ind[2]};
    transform_coord<scalar_t>(xyz, tree.offset, tree.scaling);
    scalar_t _cube_sz;
    return query_single_from_root<scalar_t>(data, tree.child,
            xyz, &_cube_sz, node_id);
}

template <typename scalar_t>
__global__ void query_single_kernel(
        PackedTreeSpec<scalar_t> tree,
        const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> indices,
        torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> values_out,
        torch::PackedTensorAccessor32<int32_t, 1, torch::RestrictPtrTraits> node_ids_out) {
    CUDA_GET_THREAD_ID(tid, indices.size(0));
    scalar_t* data_ptr = get_tree_leaf_ptr(tree.data, tree, &indices[tid][0], &node_ids_out[tid]);
    for (int i = 0; i < tree.data.size(4); ++i)
        values_out[tid][i] = data_ptr[i];
}

template <typename scalar_t>
__global__ void query_single_kernel_backward(
       PackedTreeSpec<scalar_t> tree,
       const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> indices,
       const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> grad_output,
       torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits> grad_data_out) {
    CUDA_GET_THREAD_ID(tid, indices.size(0));
    int32_t _node_id;
    scalar_t* data_ptr = get_tree_leaf_ptr(grad_data_out, tree, &indices[tid][0], &_node_id);
    for (int i = 0; i < grad_output.size(1); ++i)
        atomicAdd(&data_ptr[i], grad_output[tid][i]);
}

template <typename scalar_t>
__global__ void assign_single_kernel(
       PackedTreeSpec<scalar_t> tree,
       const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> indices,
       const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> values) {
    CUDA_GET_THREAD_ID(tid, indices.size(0));
    int32_t _node_id;
    scalar_t* data_ptr = get_tree_leaf_ptr(tree.data, tree, &indices[tid][0], &_node_id);
    for (int i = 0; i < values.size(1); ++i)
        data_ptr[i] = values[tid][i];
}

}  // namespace device
}  // namespace

QueryResult query_vertical(TreeSpec& tree, torch::Tensor indices) {
    tree.check();
    check_indices(indices);
    DEVICE_GUARD(indices);

    const auto Q = indices.size(0), K = tree.data.size(4);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);
    torch::Tensor values = torch::empty({Q, K}, indices.options());
    torch::Tensor node_ids = torch::empty({Q}, tree.child.options());
    AT_DISPATCH_FLOATING_TYPES(indices.type(), __FUNCTION__, [&] {
        device::query_single_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indices.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                values.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                node_ids.packed_accessor32<int32_t, 1, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return QueryResult(values, node_ids);
}

void assign_vertical(TreeSpec& tree, torch::Tensor indices, torch::Tensor values) {
    tree.check();
    check_indices(indices);
    check_indices(values);
    DEVICE_GUARD(indices);
    const int blocks = CUDA_N_BLOCKS_NEEDED(indices.size(0), CUDA_N_THREADS);
    AT_DISPATCH_FLOATING_TYPES(indices.type(), __FUNCTION__, [&] {
        device::assign_single_kernel<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indices.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                values.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
}

torch::Tensor query_vertical_backward(
        TreeSpec& tree,
        torch::Tensor indices,
        torch::Tensor grad_output) {
    tree.check();
    DEVICE_GUARD(indices);
    const auto Q = indices.size(0), N = tree.child.size(1),
               K = grad_output.size(1), M = tree.child.size(0);
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, CUDA_N_THREADS);

    torch::Tensor grad_data = torch::zeros({M, N, N, N, K}, grad_output.options());

    AT_DISPATCH_FLOATING_TYPES(indices.type(), __FUNCTION__, [&] {
        device::query_single_kernel_backward<scalar_t><<<blocks, CUDA_N_THREADS>>>(
                tree,
                indices.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                grad_data.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>());
    });

    CUDA_CHECK_ERRORS;
    return grad_data;
}
